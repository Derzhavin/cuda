#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */


#include "../../../../common/book.h"
#include "../../../../common/hip/hip_runtime_api.h"
#include "../../../../common/helper_string.h"
#include <sys/time.h>
#include<unistd.h>


// N blocks * 1 thread / block = N parallel thread
// N/2, 2
// N/4, 4

// int tid = threadIdx.x + blockIdx.x * blockDim.x
// gridDim



void print_device_info()
{
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess)
  {
    printf("hipGetDeviceCount returned %d\n-> %s\n", static_cast<int>(error_id), hipGetErrorString(error_id));
    printf("Result = FAIL\n");
    exit(EXIT_FAILURE);
  }

  // This function call returns 0 if there are no CUDA capable devices.
  if (deviceCount == 0)
  {
    printf("There are no available device(s) that support CUDA\n");
  }
  else
  {
    printf("Detected %d CUDA Capable device(s)\n", deviceCount);
  }

  int dev, driverVersion = 0, runtimeVersion = 0;

  for (dev = 0; dev < deviceCount; ++dev)
  {
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

    // Console log
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
        driverVersion / 1000, (driverVersion % 100) / 10,
        runtimeVersion / 1000, (runtimeVersion % 100) / 10);
    printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
        deviceProp.major, deviceProp.minor);

    char msg[256];
    snprintf(msg, sizeof(msg),
    "  Total amount of global memory:                 %.0f MBytes "
    "(%llu bytes)\n",
    static_cast<float>(deviceProp.totalGlobalMem / 1048576.0f),
    (unsigned long long)deviceProp.totalGlobalMem);

    printf("%s", msg);

    printf("  (%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
           deviceProp.multiProcessorCount,
           _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
           _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);

    printf("  GPU Max Clock rate:                            %.0f MHz (%0.2f GHz)\n",
    deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);

    #if CUDART_VERSION >= 5000
    // This is supported in CUDA 5.0 (runtime API device properties)
    printf("  Memory Clock rate:                             %.0f Mhz\n",
    deviceProp.memoryClockRate * 1e-3f);
    printf("  Memory Bus Width:                              %d-bit\n",
    deviceProp.memoryBusWidth);

    if (deviceProp.l2CacheSize)
    {
      printf("  L2 Cache Size:                                 %d bytes\n",
          deviceProp.l2CacheSize);
    }

    #else
    // This only available in CUDA 4.0-4.2 (but these were only exposed in the
    // CUDA Driver API)
    int memoryClock;
    getCudaAttribute<int>(&memoryClock, hipDeviceAttributeMemoryClockRate,
              dev);
    printf("  Memory Clock rate:                             %.0f Mhz\n",
    memoryClock * 1e-3f);
    int memBusWidth;
    getCudaAttribute<int>(&memBusWidth,
              hipDeviceAttributeMemoryBusWidth, dev);
    printf("  Memory Bus Width:                              %d-bit\n",
    memBusWidth);
    int L2CacheSize;
    getCudaAttribute<int>(&L2CacheSize, hipDeviceAttributeL2CacheSize, dev);

    if (L2CacheSize) {
    printf("  L2 Cache Size:                                 %d bytes\n",
    L2CacheSize);
    }
    #endif

    printf("  Total number of registers available per block: %d\n",
    deviceProp.regsPerBlock);
    printf("  Warp size:                                     %d\n",
    deviceProp.warpSize);
    printf("  Maximum number of threads per multiprocessor:  %d\n",
    deviceProp.maxThreadsPerMultiProcessor);
    printf("  Maximum number of threads per block:           %d\n",
    deviceProp.maxThreadsPerBlock);
    printf("  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
    deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
    deviceProp.maxThreadsDim[2]);
    printf("  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n",
    deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
    deviceProp.maxGridSize[2]);
    printf("  Integrated GPU sharing Host Memory:            %s\n",
    deviceProp.integrated ? "Yes" : "No");
  }
}

namespace gpu_add_small
{
  #define N   1024

  __global__ void add( int *a, int *b, int *c )
  {
    int tid = threadIdx.x;
    if (tid < N)
      c[tid] = a[tid] + b[tid];
  }

  void gpu_main( void )
  {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++)
    {
      a[i] = i;
      b[i] = i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice ) );

    // N/1024
    // (N+1024-1)/1024
    // ceil
    add<<<1,N>>>( dev_a, dev_b, dev_c );

    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost ) );

    // display the results
    printf( "%d + %d = %d\n", a[0], b[0], c[0] );
    printf( "%d + %d = %d\n", a[10], b[10], c[10] );
    printf( "%d + %d = %d\n", a[100], b[100], c[100] );

    // free the memory allocated on the GPU
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );
  }
}

namespace gpu_add_long
{
  #define N_2 60000


/*

block 0  |  thread 0  |  thread 1  |  thread 2  |  thread 3  |

block 1  |  thread 0  |  thread 1  |  thread 2  |  thread 3  |

block 2  |  thread 0  |  thread 1  |  thread 2  |  thread 3  |

block 3  |  thread 0  |  thread 1  |  thread 2  |  thread 3  |

*/

// N / 1024
  __global__ void add( int *a, int *b, int *c )
  {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N_2)
    {
      c[tid] = a[tid] + b[tid];
      tid += blockDim.x * gridDim.x;
    }
  }

  void gpu_main( void )
  {
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on the CPU
    a = (int*)malloc( N_2 * sizeof(int) );
    b = (int*)malloc( N_2 * sizeof(int) );
    c = (int*)malloc( N_2 * sizeof(int) );

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, N_2 * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b, N_2 * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, N_2 * sizeof(int) ) );

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N_2; i++)
    {
      a[i] = i;
      b[i] = i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N_2 * sizeof(int), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N_2 * sizeof(int), hipMemcpyHostToDevice ) );

    hipEvent_t start, stop;
    float gpuTime = 0.0;

    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 1024;
    int blocksPerGrid =(N_2 + threadsPerBlock - 1) / threadsPerBlock;
    printf("\n\nCUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    add<<<blocksPerGrid, threadsPerBlock>>>( dev_a, dev_b, dev_c );

    //add<<<128,128>>>( dev_a, dev_b, dev_c );

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &gpuTime, start, stop );

    hipEventDestroy( start );
    hipEventDestroy( stop );

    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( c, dev_c, N_2 * sizeof(int), hipMemcpyDeviceToHost ) );

    // verify that the GPU did the work we requested
    bool success = true;
    for (int i=0; i<N_2; i++)
    {
      if ((a[i] + b[i]) != c[i])
      {
        printf( "Error:  %d + %d != %d\n", a[i], b[i], c[i] );
        success = false;
      }
    }

    if (success)
    {
      printf("time on GPU = %f miliseconds\n", gpuTime);
      printf( "We did it!\n" );
      // display the results
      printf( "%d + %d = %d\n", a[0], b[0], c[0] );
      printf( "%d + %d = %d\n", a[10], b[10], c[10] );
      printf( "%d + %d = %d\n", a[50000], b[50000], c[50000] );
    }

    // free the memory we allocated on the GPU
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );

    // free the memory we allocated on the CPU
    free( a );
    free( b );
    free( c );
  }
}

int main()
{
  print_device_info();
  //gpu_add_small::gpu_main();
  gpu_add_long::gpu_main();
  return 0;
}

